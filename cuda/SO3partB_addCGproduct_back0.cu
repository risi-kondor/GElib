#include "hip/hip_runtime.h"
/*
 * This file is part of GElib, a C++/CUDA library for group equivariant 
 * tensor operations. 
 *  
 * Copyright (c) 2023, Imre Risi Kondor
 *
 * This source code file is subject to the terms of the noncommercial 
 * license distributed with GElib in the file NONCOMMERICAL.TXT. Commercial 
 * use is prohibited. All redistributed versions of this file (in orginal
 * or modified form) must retain this copyright notice and must be 
 * accompanied by a verbatim copy of the license. 
 *
 */

#ifndef _SO3partB_addCGproduct_back0_cu
#define _SO3partB_addCGproduct_back0_cu

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "SO3_CGbank.hpp"
#include "Ctensor3_view.hpp"
#include "cuda_loaders.cu"


extern GElib::SO3_CGbank SO3_cgbank;




__global__ void SO3partB_addCGproduct_back0_tiled_kernel(const cnine::Ctensor4_view_t3 x, const cnine::Ctensor3_view r, 
  const cnine::Ctensor4_view_t3 y, const int Cptr, float* cptr_global, const bool preloadCG){

  extern __shared__ unsigned char _shared[]; 
  const int b=blockIdx.x;
  const int t=threadIdx.x;

  int l1=(x.n1-1)/2;
  int l2=(y.n1-1)/2;
  int l=(r.n1-1)/2;
  int L2=y.n1;

  float* cptr;
  float* xpr;
  if(preloadCG){
    cptr=reinterpret_cast<float*>(_shared);
    xpr=cptr+((x.n1*y.n1-1)/32+1)*32;
    if(Cptr>=0) loadf(cptr,reinterpret_cast<float*>(cg_cmem)+Cptr,x.n1*y.n1);
    else loadf(cptr,cptr_global,x.n1*y.n1);
  }else{
    if(Cptr>=0) cptr=reinterpret_cast<float*>(cg_cmem)+Cptr;
    else cptr=cptr_global;
    xpr=reinterpret_cast<float*>(_shared);
  }

  float* xpi=xpr+x.n1*x.n3;
  float* ypr=xpr+((2*x.n1*x.n3-1)/32+1)*32;
  float* ypi=ypr+y.n1*y.n3;

  int xs1=x.n3;
  int ys1=y.n3;
  int rs1=r.s1;
  int ytot=(y.n2-1)*y.n3+y.last;


  for(int i=0; i<x.n2; i++){
    int xn; if(i<x.n2-1) xn=x.n3; else xn=x.last; 
    loadg_tile(xpr,x,b,i,xn);

    for(int j=0; j<y.n2; j++){
      int yn; if(j<y.n2-1) yn=y.n3; else yn=y.last;
      loadg_tile(ypr,y,b,j,yn);

      __syncthreads();

      if(t<xn){
	float* _xpr=xpr+t;
	float* _xpi=xpi+t;
    
	for(int m1=-l1; m1<=l1; m1++){
	  int lower=-l-m1; if(lower<-l2) lower=-l2;
	  int upper=l-m1; if(upper>l2) upper=l2;
	  float x_r=0;
	  float x_i=0;

	  for(int ycol=0; ycol<yn; ycol++){

	    float* _ypr=ypr+ycol;
	    float* _ypi=ypi+ycol;
	    float* _rpr=r.arr+r.s0*b+r.s2*((i*x.n3+t)*ytot+(j*y.n3+ycol));
	    float* _rpi=r.arrc+r.s0*b+r.s2*((i*x.n3+t)*ytot+(j*y.n3+ycol));

	    for(int m2=lower; m2<=upper; m2++){
	      float c=cptr[(m1+l1)*L2+m2+l2];
	      const float y_r=_ypr[ys1*(m2+l2)];
	      const float y_i=_ypi[ys1*(m2+l2)];
	      const float g_r=_rpr[rs1*(m1+m2+l)];
	      const float g_i=_rpi[rs1*(m1+m2+l)];
	      x_r+=c*(g_r*y_r+g_i*y_i);
	      x_i+=c*(-g_r*y_i+g_i*y_r);
	    }
	  }

	  _xpr[xs1*(m1+l1)]+=x_r; 
	  _xpi[xs1*(m1+l1)]+=x_i;
	}

      }// end t<xn loop
      __syncthreads();

    }// end j<y.n2 loop

    saveg_tile(xpr,x,b,i,xn);
  }// end i<x.n2 loop

}


namespace GElib{


  void SO3partB_addCGproduct_back0_cu(const cnine::Ctensor3_view& x, cnine::Ctensor3_view r, const cnine::Ctensor3_view& y, 
    const int offs, const hipStream_t& stream){

    const int xl=(x.n1-1)/2;
    const int yl=(y.n1-1)/2;
    const int l=(r.n1-1)/2;
    const int b=r.n0;

    r.arr+=r.s2*offs;
    r.arrc+=r.s2*offs;
    r.n2=x.n2*y.n2;

    float* cptr=nullptr;
    int Cptr=SO3_cgbank.getfC(xl,yl,l)/4;
    if(Cptr<0) cptr=SO3_cgbank.getf(CGindex(xl,yl,l),r.dev).arrg;
    int clines=cnine::roundup(x.n1*y.n1,32)/32;

    // set tile sizes
    const int xn=std::min(x.n2,32);
    const int yn=std::min(y.n2,32);
    cnine::Ctensor4_view_t3 xtiled(x,xn);
    cnine::Ctensor4_view_t3 ytiled(y,yn);

    int nlines=cnine::roundup(xtiled.n1*xn*2,32)/32+
      cnine::roundup(ytiled.n1*yn*2,32)/32;

    if(nlines<=384){
      bool preloadCG=(nlines+clines<=384);
      SO3partB_addCGproduct_back0_tiled_kernel<<<b,cnine::roundup(xn,32),(nlines+preloadCG*clines)*128,stream>>>
	(xtiled,r,ytiled,Cptr,cptr,preloadCG);
      return;
    }

    cout<<"error"<<endl;

  }    


}


#endif 



  /*
  if(t<32){
    int xn=xview.n1;
    int xs0=xview.s0;
    int xs1=xview.s1;
    int xarr=xview.arr;
    int xarrc=xview.arrc;
    for(int i=0; i<2*l1+1; i++)
      for(int j=0; j<xn; x++)
	xpr[i*xwidth+j]=xarr[i*xs0+j*xs1];
    for(int i=0; i<2*l1+1; i++)
      for(int j=0; j<xn; x++)
	xpi[i*xwidth+j]=xarrc[i*xs0+j*xs1];
  }

  if(t<32){
    int yn=yview.n1;
    int ys0=yview.s0;
    int ys1=yview.s1;
    int yarr=yview.arr;
    int yarrc=yview.arrc;
    for(int i=0; i<2*l2+1; i++)
      for(int j=0; j<xn; x++)
	ypr[i*ywidth+j]=yarr[i*ys0+j*ys1];
    for(int i=0; i<2*l2+1; i++)
      for(int j=0; j<xn; x++)
	ypi[i*ywidth+j]=yarrc[i*ys0+j*ys1];
  }

  if(t<rwidth){
    for(int m1=-l1; m1<=l1; m1++){
      const float x_r=xpr[xwidth*(m1+l1)];
      const float x_i=xpi[xwidth*(m1+l1)];
      int lower=-l-m1; if(lower<-l2) lower=-l2;
      int upper=l-m1; if(upper>l2) upper=l2;
      for(int m2=lower; m2<=upper; m2++){
	float c=C_ptr[(m1+l1)*r2+m2+l2];
	const float y_r=shared[ypr+ywidth*(m2+l2)];
	const float y_i=shared[ypi+ywidth*(m2+l2)];
	shared[rpr+rwidth*(m1+m2+l)]+=c*(x_r*y_r-x_i*y_i); 
	shared[rpi+rwidth*(m1+m2+l)]+=c*(x_r*y_i+x_i*y_r);
      }
    }
  }
  */
/*
__device__ int loadg1(const cnine::Ctensor3_view& x, float* dest, const int b, const int t){
  int I=x.n1;
  int J=x.n2;
  int s1=x.s1;
  int s2=x.s2;
  int offs=I*J;
  float* destc=dest+offs;
  float* source=x.arr+x.s0*b;
  float* sourcec=x.arrc+x.s0*b;
  if(t<J){
    for(int i=0; i<I; i++)
      dest[i*J+t]=source[i*s1+t*s2];
    for(int i=0; i<I; i++)
      destc[i*J+t]=sourcec[i*s1+t*s2];
  }
  return offs;
}


__device__ int saveg1(const cnine::Ctensor3_view& x, float* source, const int b, const int t){
  int I=x.n1;
  int J=x.n2;
  int s1=x.s1;
  int s2=x.s2;
  int offs=I*J; //((I*J-1)/32+1)*32;
  float* sourcec=source+offs;
  float* dest=x.arr+x.s0*b;
  float* destc=x.arrc+x.s0*b;
  if(t<J){
    for(int i=0; i<I; i++)
      dest[i*s1+t*s2]=source[i*J+t];
    for(int i=0; i<I; i++)
      destc[i*s1+t*s2]=sourcec[i*J+t];
  }
  return offs;
}
*/
/*
__global__ void SO3partB_addCGproduct_back0_kernel(const cnine::Ctensor3_view x, const cnine::Ctensor3_view r, 
  const cnine::Ctensor3_view y, const int Cptr){

  extern __shared__ unsigned char _shared[]; 
  const float* C_ptr=reinterpret_cast<float*>(cg_cmem)+Cptr;
  const int b=blockIdx.x;
  const int t=threadIdx.x;

  int l1=(x.n1-1)/2;
  int l2=(y.n1-1)/2;
  int l=(r.n1-1)/2;
  int xn=x.n2;
  int yn=y.n2;
  int rn=xn*yn;
  int L2=y.n1;

  float* xpr=reinterpret_cast<float*>(_shared);
  float* xpi=xpr+loadg(x,xpr,b,t);

  float* ypr=xpr+((2*x.n1*xn-1)/32+1)*32;
  float* ypi=ypr+loadg(y,ypr,b,t);

  float* rpr=ypr+((2*y.n1*yn-1)/32+1)*32;
  float* rpi=rpr+loadg(r,rpr,b,t);

  __syncthreads();


  float* _xpr=xpr+t;
  float* _xpi=xpi+t;

  for(int ycol=0; ycol<yn; ycol++){
    if(t<xn){

      float* _ypr=ypr+ycol;
      float* _ypi=ypi+ycol;
      
      float* _rpr=rpr+t*yn+ycol;
      float* _rpi=rpi+t*yn+ycol;
      
      for(int m1=-l1; m1<=l1; m1++){
	int lower=-l-m1; if(lower<-l2) lower=-l2;
	int upper=l-m1; if(upper>l2) upper=l2;
	for(int m2=lower; m2<=upper; m2++){
	  float c=C_ptr[(m1+l1)*L2+m2+l2];
	  const float y_r=_ypr[yn*(m2+l2)];
	  const float y_i=_ypi[yn*(m2+l2)];
	  const float g_r=_rpr[rn*(m1+m2+l)];
	  const float g_i=_rpi[rn*(m1+m2+l)];
	  _xpr[xn*(m1+l1)]+=c*(g_r*y_r+g_i*y_i);
	  _xpi[xn*(m1+l1)]+=c*(-g_r*y_i+g_i*y_r);
	}
      }
    }
    __syncthreads();
  }
  

  __syncthreads();
  
  saveg(x,xpr,b,t);

}
*/

    /*
    if(nlines<=384){
      SO3partB_addCGproduct_back0_kernel<<<b,cnine::roundup(xg.n2*y.n2,32),nlines*128,stream>>>
	(xg,rg,y,Cptr);
    }else{
      cout<<"error"<<endl;
    }
    */
