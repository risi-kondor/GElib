#include "hip/hip_runtime.h"
/*
 * This file is part of GElib, a C++/CUDA library for group equivariant 
 * tensor operations. 
 *  
 * Copyright (c) 2023, Imre Risi Kondor
 *
 * This source code file is subject to the terms of the noncommercial 
 * license distributed with GElib in the file NONCOMMERICAL.TXT. Commercial 
 * use is prohibited. All redistributed versions of this file (in orginal
 * or modified form) must retain this copyright notice and must be 
 * accompanied by a verbatim copy of the license. 
 *
 */

#ifndef _SO3partA_CGproduct_cu
#define _SO3partA_CGproduct_cu

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/complex.h>
#include <thrust/tuple.h>

//__device__ __constant__ unsigned char cg_cmem[32276]; 


#include "SO3partA.hpp"
#include "SO3partArrayA.hpp"
#include "SO3_CGbank.hpp"

#include "CellwiseBinaryCmap.hpp"
#include "BroadcastBinaryCmap.hpp"
#include "InnerCmap.hpp"
#include "OuterCmap.hpp"
#include "MVprodCmap.hpp"
#include "VMprodCmap.hpp"
//#include "convolve1_cmap.hpp"
#include "Convolve2Cmap.hpp"

extern GElib::SO3_CGbank SO3_cgbank;


__device__ void SO3part_load_lines(float* dest, const float* source, const int nlines, const int t){
  if(t<32){
    for(int i=0; i<nlines; i++)
      dest[i*32+t]=source[i*32+t];
  }
}

__device__ void SO3part_save_lines(const float* source, float* dest, const int nlines, const int t){
  if(t<32){
    for(int i=0; i<nlines; i++)
      dest[i*32+t]=source[i*32+t];
  }
}


// ---- CGproduct --------------------------------------------------------------------------------------------


template<typename IMAP>
__global__ void SO3partA_CGproduct_kernel(float* rarr, float* rarrc, float* xarr, float* xarrc, 
  float* yarr, float* yarrc, const int rstride, const int xstride, const int ystride, const IMAP cmap, 
  const int xn, const int yn, const int rn, const int l1, const int l2, const int l, 
  const int _offs, const int nch, const int Cptr, const int mode=0){

  extern __shared__ unsigned char _shared[]; 
  float* shared=reinterpret_cast<float*>(_shared);

  const float* C_ptr=reinterpret_cast<float*>(cg_cmem)+Cptr;
  const int t=threadIdx.x;

  const int r=2*l+1;
  const int r1=2*l1+1;
  const int r2=2*l2+1;

  const int xwidth=xn*nch; 
  const int ywidth=yn*nch; 
  const int rwidth=xn*yn*nch;
  const int global_rwidth=rn*nch;
  
  const int rlines=((r*rwidth-1)/32+1);
  const int xlines=((r1*xwidth-1)/32+1);
  const int ylines=((r2*ywidth-1)/32+1);

  const int rptr=0;
  const int xptr=rptr+rlines*64;
  const int yptr=xptr+xlines*64;

  int rix,xix,yix;
  int nsum;
  int lst;


  if(mode<2){
    auto T=cmap(blockIdx.x,blockIdx.y,blockIdx.z);
    rix=thrust::get<0>(T);
    xix=thrust::get<1>(T);
    yix=thrust::get<2>(T);
    nsum=1;
    //if(t==0) printf("foop1\n");
  }else{
    rix=cmap.target(blockIdx.x);
    nsum=cmap.n_accum(blockIdx.x);
    lst=cmap.lst_ptr(blockIdx.x);
  }
  
  if(mode==1){
    if(t<32){
      for(int i=0; i<2*rlines; i++)
	shared[rptr+i*32+t]=0;
    }
  }else{
    if(t<rwidth){
      for(int i=0; i<r; i++)
	shared[rptr+i*rwidth+t]=rarr[rix*rstride+_offs+i*global_rwidth+t];
      for(int i=0; i<r; i++)
	shared[rptr+rlines*32+i*rwidth+t]=rarrc[rix*rstride+_offs+i*global_rwidth+t];
    }
  }
  
  for(int s=0; s<nsum; s++){

    if(mode==2){
      auto T=cmap.source(lst,blockIdx.x,s);
      xix=thrust::get<0>(T);
      yix=thrust::get<1>(T);
    }

    SO3part_load_lines(shared+xptr,xarr+xix*xstride,xlines,t);
    SO3part_load_lines(shared+xptr+xlines*32,xarrc+xix*xstride,xlines,t);
    SO3part_load_lines(shared+yptr,yarr+yix*ystride,ylines,t);
    SO3part_load_lines(shared+yptr+ylines*32,yarrc+yix*ystride,ylines,t);

    //if(t==0) printf("foop3\n");
      
      __syncthreads();

      const int rpr=rptr+t;
      const int rpi=rpr+rlines*32;

      const int xcol=t/yn;
      const int xpr=xptr+xcol;
      const int xpi=xpr+xlines*32;

      const int ycol=t%ywidth;
      const int ypr=yptr+ycol;
      const int ypi=ypr+ylines*32;


      if(t<rwidth){
	for(int m1=-l1; m1<=l1; m1++){
	  const float x_r=shared[xpr+xwidth*(m1+l1)];
	  const float x_i=shared[xpi+xwidth*(m1+l1)];
	  int lower=-l-m1; if(lower<-l2) lower=-l2;
	  int upper=l-m1; if(upper>l2) upper=l2;
	  for(int m2=lower; m2<=upper; m2++){
	    float c=C_ptr[(m1+l1)*r2+m2+l2];
	    const float y_r=shared[ypr+ywidth*(m2+l2)];
	    const float y_i=shared[ypi+ywidth*(m2+l2)];
	    shared[rpr+rwidth*(m1+m2+l)]+=c*(x_r*y_r-x_i*y_i); 
	    shared[rpi+rwidth*(m1+m2+l)]+=c*(x_r*y_i+x_i*y_r);
	  }
	}
      }

    //if(t==0) printf("foop4\n");

      __syncthreads();
  }

  //if(t==0) printf("fooq\n");
  
  if(t<rwidth){
    for(int i=0; i<r; i++)
      rarr[rix*rstride+_offs+i*global_rwidth+t]=shared[rptr+i*rwidth+t];
    for(int i=0; i<r; i++)
      rarrc[rix*rstride+_offs+i*global_rwidth+t]=shared[rptr+rlines*32+i*rwidth+t];
  }    
 
}



template<typename IMAP>
__global__ void SO3partA_CGproduct_kernel_L(float* rarr, float* rarrc, float* xarr, float* xarrc, 
  float* yarr, float* yarrc, const int rstride, const int xstride, const int ystride, const IMAP cmap, 
  const int xn, const int yn, const int rn, const int l1, const int l2, const int l, 
  const int _offs, const int nch, const int Cptr, const int mode=0){

  extern __shared__ unsigned char _shared[]; 
  float* shared=reinterpret_cast<float*>(_shared);

  const float* C_ptr=reinterpret_cast<float*>(cg_cmem)+Cptr;
  const int t=threadIdx.x;

  const int r=2*l+1;
  const int r1=2*l1+1;
  const int r2=2*l2+1;

  const int xwidth=xn*nch; 
  const int ywidth=yn*nch; 
  const int rwidth=xn*nch; 
  const int global_rwidth=rn*nch;
  
  const int rlines=((r*rwidth-1)/32+1);
  const int xlines=((r1*xwidth-1)/32+1);
  const int ylines=((r2*1-1)/32+1);

  const int rptr=0;
  const int xptr=rptr+rlines*64;
  const int yptr=xptr+xlines*64;

  int rix,xix,yix;
  int nsum;
  int lst;

  if(mode<2){
    auto T=cmap(blockIdx.x,blockIdx.y,blockIdx.z);
    rix=thrust::get<0>(T);
    xix=thrust::get<1>(T);
    yix=thrust::get<2>(T);
    nsum=1;
  }else{
    rix=cmap.target(blockIdx.x);
    nsum=cmap.n_accum(blockIdx.x);
    lst=cmap.lst_ptr(blockIdx.x);
  }
  

  for(int s=0; s<nsum; s++){

    if(mode==2){
      auto T=cmap.source(lst,blockIdx.x,s);
      xix=thrust::get<0>(T);
      yix=thrust::get<1>(T);
    }

    SO3part_load_lines(shared+xptr,xarr+xix*xstride,xlines,t);
    SO3part_load_lines(shared+xptr+xlines*32,xarrc+xix*xstride,xlines,t);


    for(int ycol=0; ycol<ywidth; ycol++){

      if(t==0){
	for(int i=0; i<r; i++)
	  shared[yptr+i]=yarr[yix*ystride+i*ywidth+ycol];
	for(int i=0; i<r; i++)
	  shared[yptr+ylines*32+i]=yarrc[yix*ystride+i*ywidth+ycol];
      }

      if(t<32){
	for(int i=0; i<2*rlines; i++)
	  shared[rptr+i*32+t]=0;
      }
      
      __syncthreads();

      const int rpr=rptr+t;
      const int rpi=rpr+rlines*32;

      const int xcol=t;
      const int xpr=xptr+xcol;
      const int xpi=xpr+xlines*32;

      //const int ycol=t%ywidth;
      const int ypr=yptr;// +ycol;
      const int ypi=ypr+ylines*32;


      if(t<rwidth){
	for(int m1=-l1; m1<=l1; m1++){
	  const float x_r=shared[xpr+xwidth*(m1+l1)];
	  const float x_i=shared[xpi+xwidth*(m1+l1)];
	  int lower=-l-m1; if(lower<-l2) lower=-l2;
	  int upper=l-m1; if(upper>l2) upper=l2;
	  for(int m2=lower; m2<=upper; m2++){
	    float c=C_ptr[(m1+l1)*r2+m2+l2];
	    const float y_r=shared[ypr+1*(m2+l2)];
	    const float y_i=shared[ypi+1*(m2+l2)];
	    shared[rpr+rwidth*(m1+m2+l)]+=c*(x_r*y_r-x_i*y_i); 
	    shared[rpi+rwidth*(m1+m2+l)]+=c*(x_r*y_i+x_i*y_r);
	  }
	}
      }

      //if(t==0) printf("foop4\n");

      __syncthreads();

      //if(t==0) printf("fooq\n");
  
      if(t<rwidth){
	for(int i=0; i<r; i++)
	  rarr[rix*rstride+_offs+i*global_rwidth+t*ywidth+ycol]+=shared[rptr+i*rwidth+t];
	for(int i=0; i<r; i++)
	  rarrc[rix*rstride+_offs+i*global_rwidth+t*ywidth+ycol]+=shared[rptr+rlines*32+i*rwidth+t];
      }    

      __syncthreads();

    } // ycol

  } //nsum
}




// ---- back0 ------------------------------------------------------------------------------------------------


template<typename IMAP>
__global__ void SO3partA_CGproduct_back0_kernel(float* xarr, float* xarrc, float* garr, float* garrc, 
  float* yarr, float* yarrc, const int xstride, const int ystride, const int gstride, const IMAP cmap, 
  const int xn, const int yn, const int gn, const int l1, const int l2, const int l, 
  const int _offs, const int nch, const int Cptr, const int mode=0){

  extern __shared__ unsigned char _shared[]; 
  float* shared=reinterpret_cast<float*>(_shared);

  const float* C_ptr=reinterpret_cast<float*>(cg_cmem)+Cptr;
  const int t=threadIdx.x;

  const int rg=2*l+1;
  const int rx=2*l1+1;
  const int ry=2*l2+1;

  const int xwidth=xn*nch; 
  const int ywidth=yn*nch; 
  const int gwidth=xn*yn*nch;
  const int global_gwidth=gn*nch;

  const int glines=((rg*gwidth-1)/32+1);
  const int xlines=((rx*xwidth-1)/32+1);
  const int ylines=((ry*ywidth-1)/32+1);

  const int xptr=0;
  const int gptr=xptr+xlines*64;
  const int yptr=gptr+glines*64;

  int gix,xix,yix;
  int nsum;
  int lst;

  if(mode<2){
    auto T=cmap(blockIdx.x,blockIdx.y,blockIdx.z);
    xix=thrust::get<0>(T);
    gix=thrust::get<1>(T);
    yix=thrust::get<2>(T);
    nsum=1;
  }else{
    xix=cmap.target(blockIdx.x);
    nsum=cmap.n_accum(blockIdx.x);
    lst=cmap.lst_ptr(blockIdx.x);
  }

  if(mode==1){
    if(t<32){
      for(int i=0; i<2*xlines; i++){
	shared[xptr+i*32+t]=0;
      }
    }
  }else{
    SO3part_load_lines(shared+xptr,xarr+xix*xstride,xlines,t);
    SO3part_load_lines(shared+xptr+xlines*32,xarrc+xix*xstride,xlines,t);
  }
  
  for(int s=0; s<nsum; s++){

    if(mode==2){
      auto T=cmap.source(lst,blockIdx.x,s);
      gix=thrust::get<0>(T);
      yix=thrust::get<1>(T);
    }

    // hack: gwidth assumed to be <=32
    for(int i=0; i<rg; i++)
      if(t<gwidth)
	shared[gptr+i*gwidth+t]=garr[gix*gstride+i*global_gwidth+_offs+t];
    for(int i=0; i<rg; i++)
      if(t<gwidth)
	shared[gptr+glines*32+i*gwidth+t]=garrc[gix*gstride+i*global_gwidth+_offs+t];

    SO3part_load_lines(shared+yptr,yarr+yix*ystride,ylines,t);
    SO3part_load_lines(shared+yptr+ylines*32,yarrc+yix*ystride,ylines,t);

    __syncthreads();

      //const int xcol=t;
    const int xpr=xptr+t;
    const int xpi=xpr+xlines*32;
    
    for(int ycol=0; ycol<ywidth; ycol++){
      
      const int ypr=yptr+ycol;
      const int ypi=ypr+ylines*32;
      
      const int gpr=gptr+ywidth*t+ycol;
      const int gpi=gpr+glines*32;

      if(t<xwidth){
	for(int m1=-l1; m1<=l1; m1++){
	  int lower=-l-m1; if(lower<-l2) lower=-l2;
	  int upper=l-m1; if(upper>l2) upper=l2;
	  for(int m2=lower; m2<=upper; m2++){
	    float c=C_ptr[(m1+l1)*ry+m2+l2];
	    const float y_r=shared[ypr+ywidth*(m2+l2)];
	    const float y_i=shared[ypi+ywidth*(m2+l2)];
	    const float g_r=shared[gpr+gwidth*(m1+m2+l)];
	    const float g_i=shared[gpi+gwidth*(m1+m2+l)];
	    shared[xpr+xwidth*(m1+l1)]+=c*(g_r*y_r+g_i*y_i); 
	    shared[xpi+xwidth*(m1+l1)]+=c*(-g_r*y_i+g_i*y_r);
	  }
	}
      }
      __syncthreads();
	
    }

  }
  
  SO3part_save_lines(shared+xptr,xarr+xix*xstride,xlines,t);
  SO3part_save_lines(shared+xptr+xlines*32,xarrc+xix*xstride,xlines,t);
    
  __syncthreads();

}


template<typename IMAP>
__global__ void SO3partA_CGproduct_back0_kernel_big(float* xarr, float* xarrc, float* garr, float* garrc, 
  float* yarr, float* yarrc, const int xstride, const int ystride, const int gstride, const IMAP cmap, 
  const int xn, const int yn, const int gn, const int l1, const int l2, const int l, 
  const int _offs, const int nch, const int Cptr, const int mode=0){

  extern __shared__ unsigned char _shared[]; 
  float* shared=reinterpret_cast<float*>(_shared);

  const float* C_ptr=reinterpret_cast<float*>(cg_cmem)+Cptr;
  const int t=threadIdx.x;

  //const int rg=2*l+1;
  const int rx=2*l1+1;
  const int ry=2*l2+1;

  const int xwidth=xn*nch; 
  const int ywidth=yn*nch; 
  //const int gwidth=xn*yn*nch;
  const int global_gwidth=gn*nch;

  //const int glines=((rg*gwidth-1)/32+1);
  const int xlines=((rx*xwidth-1)/32+1);
  const int ylines=((ry*ywidth-1)/32+1);

  const int xptr=0;
  const int yptr=xptr+xlines*64;
  //const int yptr=gptr+glines*64;

  int gix,xix,yix;
  int nsum;
  int lst;

  if(mode<2){
    auto T=cmap(blockIdx.x,blockIdx.y,blockIdx.z);
    xix=thrust::get<0>(T);
    gix=thrust::get<1>(T);
    yix=thrust::get<2>(T);
    nsum=1;
  }else{
    xix=cmap.target(blockIdx.x);
    nsum=cmap.n_accum(blockIdx.x);
    lst=cmap.lst_ptr(blockIdx.x);
  }

  if(mode==1){
    if(t<32){
      for(int i=0; i<2*xlines; i++){
	shared[xptr+i*32+t]=0;
      }
    }
  }else{
    SO3part_load_lines(shared+xptr,xarr+xix*xstride,xlines,t);
    SO3part_load_lines(shared+xptr+xlines*32,xarrc+xix*xstride,xlines,t);
  }
  
  for(int s=0; s<nsum; s++){

    if(mode==2){
      auto T=cmap.source(lst,blockIdx.x,s);
      gix=thrust::get<0>(T);
      yix=thrust::get<1>(T);
    }

    // hack: gwidth assumed to be <=32
    //for(int i=0; i<rg; i++)
    //if(t<gwidth)
    //shared[gptr+i*gwidth+t]=garr[gix*gstride+i*global_gwidth+_offs+t];
    //for(int i=0; i<rg; i++)
    //if(t<gwidth)
    //shared[gptr+glines*32+i*gwidth+t]=garrc[gix*gstride+i*global_gwidth+_offs+t];

    SO3part_load_lines(shared+yptr,yarr+yix*ystride,ylines,t);
    SO3part_load_lines(shared+yptr+ylines*32,yarrc+yix*ystride,ylines,t);

    __syncthreads();

      //const int xcol=t;
    const int xpr=xptr+t;
    const int xpi=xpr+xlines*32;
    
    for(int ycol=0; ycol<ywidth; ycol++){
      
      const int ypr=yptr+ycol;
      const int ypi=ypr+ylines*32;
      
      //const int gpr=gptr+ywidth*t+ycol;
      //const int gpi=gpr+glines*32;

      if(t<xwidth){
	for(int m1=-l1; m1<=l1; m1++){
	  int lower=-l-m1; if(lower<-l2) lower=-l2;
	  int upper=l-m1; if(upper>l2) upper=l2;
	  for(int m2=lower; m2<=upper; m2++){
	    float c=C_ptr[(m1+l1)*ry+m2+l2];
	    const float y_r=shared[ypr+ywidth*(m2+l2)];
	    const float y_i=shared[ypi+ywidth*(m2+l2)];
	    //const float g_r=shared[gpr+gwidth*(m1+m2+l)];
	    //const float g_i=shared[gpi+gwidth*(m1+m2+l)];
	    const float g_r=garr[gix*gstride+_offs+ywidth*t+ycol+(m1+m2+l)*global_gwidth];
	    const float g_i=garrc[gix*gstride+_offs+ywidth*t+ycol+(m1+m2+l)*global_gwidth];
	    shared[xpr+xwidth*(m1+l1)]+=c*(g_r*y_r+g_i*y_i); 
	    shared[xpi+xwidth*(m1+l1)]+=c*(-g_r*y_i+g_i*y_r);
	  }
	}
      }
      __syncthreads();
	
    }

  }
  
  SO3part_save_lines(shared+xptr,xarr+xix*xstride,xlines,t);
  SO3part_save_lines(shared+xptr+xlines*32,xarrc+xix*xstride,xlines,t);
    
  __syncthreads();

}


// ---- back1 ------------------------------------------------------------------------------------------------


template<typename IMAP>
__global__ void SO3partA_CGproduct_back1_kernel(float* yarr, float* yarrc, float* garr, float* garrc, 
  float* xarr, float* xarrc, const int xstride, const int ystride, const int gstride, const IMAP cmap, 
  const int xn, const int yn, const int gn, const int l1, const int l2, const int l, 
  const int _offs, const int nch, const int Cptr, const int mode=0){

  extern __shared__ unsigned char _shared[]; 
  float* shared=reinterpret_cast<float*>(_shared);

  const float* C_ptr=reinterpret_cast<float*>(cg_cmem)+Cptr;
  const int t=threadIdx.x;

  const int rg=2*l+1;
  const int rx=2*l1+1;
  const int ry=2*l2+1;

  const int xwidth=xn*nch; 
  const int ywidth=yn*nch; 
  const int gwidth=xn*yn*nch;
  const int global_gwidth=gn*nch;

  const int glines=((rg*gwidth-1)/32+1);
  const int xlines=((rx*xwidth-1)/32+1);
  const int ylines=((ry*ywidth-1)/32+1);

  const int yptr=0;
  const int gptr=yptr+ylines*64;
  const int xptr=gptr+glines*64;

  int gix,xix,yix;
  int nsum;
  int lst;

  if(mode<2){
    auto T=cmap(blockIdx.x,blockIdx.y,blockIdx.z);
    yix=thrust::get<0>(T);
    gix=thrust::get<1>(T);
    xix=thrust::get<2>(T);
    nsum=1;
  }else{
    yix=cmap.target(blockIdx.x);
    nsum=cmap.n_accum(blockIdx.x);
    lst=cmap.lst_ptr(blockIdx.x);
  }

  if(mode==1){
    if(t<32){
      for(int i=0; i<2*ylines; i++)
	shared[yptr+i*32+t]=0;
    }
  }else{
    SO3part_load_lines(shared+yptr,yarr+yix*ystride,ylines,t);
    SO3part_load_lines(shared+yptr+ylines*32,yarrc+yix*ystride,ylines,t);
  }
  
  for(int s=0; s<nsum; s++){

    if(mode==2){
      auto T=cmap.source(lst,blockIdx.x,s);
      gix=thrust::get<0>(T);
      xix=thrust::get<1>(T);
    }

    // hack: gwidth assumed to be <=32
    for(int i=0; i<rg; i++)
      if(t<gwidth)
	shared[gptr+i*gwidth+t]=garr[gix*gstride+i*global_gwidth+_offs+t];
    for(int i=0; i<rg; i++)
      if(t<gwidth)
	shared[gptr+glines*32+i*gwidth+t]=garrc[gix*gstride+i*global_gwidth+_offs+t];

    SO3part_load_lines(shared+xptr,xarr+xix*xstride,xlines,t);
    SO3part_load_lines(shared+xptr+xlines*32,xarrc+xix*xstride,xlines,t);

    __syncthreads();

    //const int ycol=t;
    const int ypr=yptr+t;
    const int ypi=ypr+ylines*32;
    
    for(int xcol=0; xcol<xwidth; xcol++){
      
      const int xpr=xptr+xcol;
      const int xpi=xpr+xlines*32;
      
      const int gpr=gptr+ywidth*xcol+t;
      const int gpi=gpr+glines*32;

      if(t<ywidth){
	for(int m1=-l1; m1<=l1; m1++){
	  int lower=-l-m1; if(lower<-l2) lower=-l2;
	  int upper=l-m1; if(upper>l2) upper=l2;
	  const float x_r=shared[xpr+xwidth*(m1+l1)];
	  const float x_i=shared[xpi+xwidth*(m1+l1)];
	  for(int m2=lower; m2<=upper; m2++){
	    float c=C_ptr[(m1+l1)*ry+m2+l2];
	    const float g_r=shared[gpr+gwidth*(m1+m2+l)];
	    const float g_i=shared[gpi+gwidth*(m1+m2+l)];
	    shared[ypr+ywidth*(m2+l2)]+=c*(g_r*x_r+g_i*x_i); 
	    shared[ypi+ywidth*(m2+l2)]+=c*(-g_r*x_i+g_i*x_r);
	  }
	}
      }
      __syncthreads();

    }

  }
  
  SO3part_save_lines(shared+yptr,yarr+yix*ystride,ylines,t);
  SO3part_save_lines(shared+yptr+ylines*32,yarrc+yix*ystride,ylines,t);
    
  __syncthreads();

}


template<typename IMAP>
__global__ void SO3partA_CGproduct_back1_kernel_big(float* yarr, float* yarrc, float* garr, float* garrc, 
  float* xarr, float* xarrc, const int xstride, const int ystride, const int gstride, const IMAP cmap, 
  const int xn, const int yn, const int gn, const int l1, const int l2, const int l, 
  const int _offs, const int nch, const int Cptr, const int mode=0){

  extern __shared__ unsigned char _shared[]; 
  float* shared=reinterpret_cast<float*>(_shared);

  const float* C_ptr=reinterpret_cast<float*>(cg_cmem)+Cptr;
  const int t=threadIdx.x;

  //const int rg=2*l+1;
  const int rx=2*l1+1;
  const int ry=2*l2+1;

  const int xwidth=xn*nch; 
  const int ywidth=yn*nch; 
  //const int gwidth=xn*yn*nch;
  const int global_gwidth=gn*nch;

  //const int glines=((rg*gwidth-1)/32+1);
  const int xlines=((rx*xwidth-1)/32+1);
  const int ylines=((ry*ywidth-1)/32+1);

  const int yptr=0;
  const int xptr=yptr+ylines*64;
  //const int xptr=gptr+glines*64;

  int gix,xix,yix;
  int nsum;
  int lst;

  if(mode<2){
    auto T=cmap(blockIdx.x,blockIdx.y,blockIdx.z);
    yix=thrust::get<0>(T);
    gix=thrust::get<1>(T);
    xix=thrust::get<2>(T);
    nsum=1;
  }else{
    yix=cmap.target(blockIdx.x);
    nsum=cmap.n_accum(blockIdx.x);
    lst=cmap.lst_ptr(blockIdx.x);
  }

  if(mode==1){
    if(t<32){
      for(int i=0; i<2*ylines; i++)
	shared[yptr+i*32+t]=0;
    }
  }else{
    SO3part_load_lines(shared+yptr,yarr+yix*ystride,ylines,t);
    SO3part_load_lines(shared+yptr+ylines*32,yarrc+yix*ystride,ylines,t);
  }
  
  for(int s=0; s<nsum; s++){

    if(mode==2){
      auto T=cmap.source(lst,blockIdx.x,s);
      gix=thrust::get<0>(T);
      xix=thrust::get<1>(T);
    }

    // hack: gwidth assumed to be <=32
    //for(int i=0; i<rg; i++)
    //if(t<gwidth)
    //shared[gptr+i*gwidth+t]=garr[gix*gstride+i*global_gwidth+_offs+t];
    //for(int i=0; i<rg; i++)
    //if(t<gwidth)
    //shared[gptr+glines*32+i*gwidth+t]=garrc[gix*gstride+i*global_gwidth+_offs+t];

    SO3part_load_lines(shared+xptr,xarr+xix*xstride,xlines,t);
    SO3part_load_lines(shared+xptr+xlines*32,xarrc+xix*xstride,xlines,t);

    __syncthreads();

    //const int ycol=t;
    const int ypr=yptr+t;
    const int ypi=ypr+ylines*32;
    
    for(int xcol=0; xcol<xwidth; xcol++){
      
      const int xpr=xptr+xcol;
      const int xpi=xpr+xlines*32;
      
      //const int gpr=gptr+ywidth*xcol+t;
      //const int gpi=gpr+glines*32;

      if(t<ywidth){
	for(int m1=-l1; m1<=l1; m1++){
	  int lower=-l-m1; if(lower<-l2) lower=-l2;
	  int upper=l-m1; if(upper>l2) upper=l2;
	  const float x_r=shared[xpr+xwidth*(m1+l1)];
	  const float x_i=shared[xpi+xwidth*(m1+l1)];
	  for(int m2=lower; m2<=upper; m2++){
	    float c=C_ptr[(m1+l1)*ry+m2+l2];
	    //const float g_r=shared[gpr+gwidth*(m1+m2+l)];
	    //const float g_i=shared[gpi+gwidth*(m1+m2+l)];
	    const float g_r=garr[gix*gstride+_offs+ywidth*xcol+t+(m1+m2+l)*global_gwidth];
	    const float g_i=garrc[gix*gstride+_offs+ywidth*xcol+t+(m1+m2+l)*global_gwidth];
	    shared[ypr+ywidth*(m2+l2)]+=c*(g_r*x_r+g_i*x_i); 
	    shared[ypi+ywidth*(m2+l2)]+=c*(-g_r*x_i+g_i*x_r);
	  }
	}
      }
      __syncthreads();

    }

  }
  
  SO3part_save_lines(shared+yptr,yarr+yix*ystride,ylines,t);
  SO3part_save_lines(shared+yptr+ylines*32,yarrc+yix*ystride,ylines,t);
    
  __syncthreads();

}


// -----------------------------------------------------------------------------------------------------------


namespace GElib{


  template<typename CMAP>
  void SO3partA_CGproduct_cu(const CMAP& map, SO3partArrayA& r, const SO3partArrayA& x, 
    const SO3partArrayA& y, const hipStream_t& stream, const int offs, const int mode){

    const int xl=x.getl();
    const int yl=y.getl();
    const int l=r.getl();
    const int _nch=1;
    assert(x.nbu==r.nbu);
    assert(y.nbu==r.nbu);
    int _nbu=1; if(_nbu<0) _nbu=1;

    int Cptr=SO3_cgbank.getfC(xl,yl,l)/4;
    //int nlines=x.cellstride/16+y.cellstride/16+r.cellstride/16; // should be smaller than this!
    int nlines=x.cellstride/16+y.cellstride/16+cnine::roundup(x.getn()*y.getn()*_nch*(2*l+1),32)/16;
    // nlines/=_nbu;

    cout<<"nlines="<<nlines<<endl;

    if(nlines<=0*384){

      SO3partA_CGproduct_kernel<<<map.blockdims(),cnine::roundup(x.getn()*y.getn(),32),nlines*128,stream>>>
	(r.arrg,r.arrgc,x.arrg,x.arrgc,y.arrg,y.arrgc,
	  r.cellstride,x.cellstride,y.cellstride,map,
	  x.getn(),y.getn(),r.getn(),xl,yl,l,offs,_nch,Cptr,mode);

    }else{
      
      int nlines=x.cellstride/16+cnine::roundup(_nch*(2*yl+1),32)/16+cnine::roundup(x.getn()*_nch*(2*l+1),32)/16;

      cout<<"GElib: large CGproduct"<<endl; 

      if(nlines>384){
	cout<<"GElib error: CGproduct too big for shared memory"<<endl;
      }else{
	SO3partA_CGproduct_kernel_L<<<map.blockdims(),cnine::roundup(x.getn(),32),nlines*128,stream>>>
	  (r.arrg,r.arrgc,x.arrg,x.arrgc,y.arrg,y.arrgc,
	    r.cellstride,x.cellstride,y.cellstride,map,
	    x.getn(),y.getn(),r.getn(),xl,yl,l,offs,_nch,Cptr,mode);
      }
    }

  }

  
  void SO3partA_CGproduct_cu(SO3partA& r, const SO3partA& x, const SO3partA& y,  const int offs, 
    const hipStream_t& stream,const int mode){

    const int xl=x.getl();
    const int yl=y.getl();
    const int l=r.getl();
    const int _nch=1;
    assert(x.nbu==r.nbu);
    assert(y.nbu==r.nbu);
    int _nbu=1; if(_nbu<0) _nbu=1;
    cnine::CellwiseBinaryCmap map;

    int Cptr=SO3_cgbank.getfC(xl,yl,l)/4;
    int nlines=cnine::roundup(x.memsize,32)/32+cnine::roundup(y.memsize,32)/32+
      cnine::roundup(x.getn()*y.getn()*_nch*(2*l+1),32)/16;

    //cout<<"nlines="<<nlines<<endl;

    if(nlines<=384){

      SO3partA_CGproduct_kernel<<<map.blockdims(),cnine::roundup(x.getn()*y.getn(),32),nlines*128,stream>>>
	(r.arrg,r.arrgc,x.arrg,x.arrgc,y.arrg,y.arrgc,
	  0,0,0,map,
	  x.getn(),y.getn(),r.getn(),xl,yl,l,offs,_nch,Cptr,mode);

    }else{
      
      int nlines=cnine::roundup(x.memsize,32)/32+cnine::roundup(y.memsize,32)/32+
        cnine::roundup(x.getn()*_nch*(2*l+1),32)/16;

      cout<<"GElib: large CGproduct"<<endl; 

      if(nlines>384){
	cout<<"GElib error: CGproduct too big for shared memory"<<endl;
      }else{
	SO3partA_CGproduct_kernel_L<<<map.blockdims(),cnine::roundup(x.getn(),32),nlines*128,stream>>>
	  (r.arrg,r.arrgc,x.arrg,x.arrgc,y.arrg,y.arrgc,
	    0,0,0,map,
	    x.getn(),y.getn(),r.getn(),xl,yl,l,offs,_nch,Cptr,mode);
      }
    }

  }

  
  template<typename CMAP>
  void SO3partA_CGproduct_back0_cu(const CMAP& map, SO3partArrayA& x, const SO3partArrayA& g, 
    const SO3partArrayA& y, const hipStream_t& stream, const int offs, const int mode){

    const int xl=x.getl();
    const int yl=y.getl();
    const int l=g.getl();

    int Cptr=SO3_cgbank.getfC(xl,yl,l)/4;
    int nlines=x.cellstride/16+y.cellstride/16+g.cellstride/16;
    assert(x.nbu==g.nbu);
    assert(y.nbu==g.nbu);

    const int _nch=1;
    int _nbu=1; if(_nbu<0) _nbu=1;
    nlines/=_nbu;

    cout<<"nlines="<<nlines<<endl;

    if(nlines<=0*384){

      SO3partA_CGproduct_back0_kernel<<<map.blockdims(),cnine::roundup(x.getn(),32),nlines*128,stream>>>
	(x.arrg,x.arrgc,g.arrg,g.arrgc,y.arrg,y.arrgc,
	  x.cellstride,y.cellstride,g.cellstride,map,
	  x.getn(),y.getn(),g.getn(),xl,yl,l,offs,_nch,Cptr,mode);
      
    }else{

      int nlines=x.cellstride/16+y.cellstride/16;
      
      cout<<"GElib: large CGproduct_back0"<<endl; 

      if(nlines>384){
	cout<<"GElib error: CGproduct too big for shared memory"<<endl;
      }else{
	SO3partA_CGproduct_back0_kernel_big<<<map.blockdims(),cnine::roundup(std::max(x.getn(),y.getn()),32),nlines*128,stream>>>
	  (x.arrg,x.arrgc,g.arrg,g.arrgc,y.arrg,y.arrgc,
	    x.cellstride,y.cellstride,g.cellstride,map,
	    x.getn(),y.getn(),g.getn(),xl,yl,l,offs,_nch,Cptr,mode);
      }

    }

  }
  

  template<typename CMAP>
  void SO3partA_CGproduct_back1_cu(const CMAP& map, SO3partArrayA& y, const SO3partArrayA& g, 
    const SO3partArrayA& x, const hipStream_t& stream, const int offs, const int mode){

    const int xl=x.getl();
    const int yl=y.getl();
    const int l=g.getl();

    int Cptr=SO3_cgbank.getfC(xl,yl,l)/4;
    int nlines=x.cellstride/16+y.cellstride/16+g.cellstride/16;
    assert(x.nbu==g.nbu);
    assert(y.nbu==g.nbu);

    const int _nch=1;
    int _nbu=1; if(_nbu<0) _nbu=1;
    nlines/=_nbu;

    cout<<"nlines="<<nlines<<endl;

    if(nlines<=0*384){

      SO3partA_CGproduct_back1_kernel<<<map.blockdims(),cnine::roundup(y.getn(),32),nlines*128,stream>>>
	(y.arrg,y.arrgc,g.arrg,g.arrgc,x.arrg,x.arrgc,
	  x.cellstride,y.cellstride,g.cellstride,map,
	  x.getn(),y.getn(),g.getn(),xl,yl,l,offs,_nch,Cptr,mode);
      
    }else{

      int nlines=x.cellstride/16+y.cellstride/16;
      
      cout<<"GElib: large CGproduct_back1"<<endl; 

      if(nlines>384){
	cout<<"GElib error: CGproduct too big for shared memory"<<endl;
      }else{
	SO3partA_CGproduct_back1_kernel_big<<<map.blockdims(),cnine::roundup(std::max(x.getn(),y.getn()),32),nlines*128,stream>>>
	  (y.arrg,y.arrgc,g.arrg,g.arrgc,x.arrg,x.arrgc,
	    x.cellstride,y.cellstride,g.cellstride,map,
	    x.getn(),y.getn(),g.getn(),xl,yl,l,offs,_nch,Cptr,mode);
      }

    }

  }


  template void SO3partA_CGproduct_cu(const cnine::CellwiseBinaryCmap& map, 
    SO3partArrayA&, const SO3partArrayA&, const SO3partArrayA&, const hipStream_t&, const int offs, 
    const int mode);

  template void SO3partA_CGproduct_cu(const cnine::BroadcastBinaryCmap& map, 
    SO3partArrayA&, const SO3partArrayA&, const SO3partArrayA&, const hipStream_t&, const int offs, 
    const int mode);

  template void SO3partA_CGproduct_cu(const cnine::OuterCmap& map, 
    SO3partArrayA&, const SO3partArrayA&, const SO3partArrayA&, const hipStream_t&, const int offs, 
    const int mode);

  template void SO3partA_CGproduct_cu(const cnine::InnerCmap& map, 
    SO3partArrayA&, const SO3partArrayA&, const SO3partArrayA&, const hipStream_t&, const int offs, 
    const int mode);

  template void SO3partA_CGproduct_cu(const cnine::MVprodCmap& map, 
    SO3partArrayA&, const SO3partArrayA&, const SO3partArrayA&, const hipStream_t&, const int offs, 
    const int mode);

  template void SO3partA_CGproduct_cu(const cnine::Convolve2Cmap& map, 
    SO3partArrayA&, const SO3partArrayA&, const SO3partArrayA&, const hipStream_t&, const int offs, 
    const int mode);



  template void SO3partA_CGproduct_back0_cu(const cnine::CellwiseBinaryCmap& map, 
    SO3partArrayA&, const SO3partArrayA&, const SO3partArrayA&, const hipStream_t&, const int offs, 
    const int mode);

  template void SO3partA_CGproduct_back0_cu(const cnine::BroadcastBinaryCmap& map, 
    SO3partArrayA&, const SO3partArrayA&, const SO3partArrayA&, const hipStream_t&, const int offs, 
    const int mode);

  template void SO3partA_CGproduct_back0_cu(const cnine::OuterCmap& map, 
    SO3partArrayA&, const SO3partArrayA&, const SO3partArrayA&, const hipStream_t&, const int offs, 
    const int mode);


  template void SO3partA_CGproduct_back1_cu(const cnine::CellwiseBinaryCmap& map, 
    SO3partArrayA&, const SO3partArrayA&, const SO3partArrayA&, const hipStream_t&, const int offs, 
    const int mode);

  template void SO3partA_CGproduct_back1_cu(const cnine::BroadcastBinaryCmap& map, 
    SO3partArrayA&, const SO3partArrayA&, const SO3partArrayA&, const hipStream_t&, const int offs, 
    const int mode);

  template void SO3partA_CGproduct_back1_cu(const cnine::OuterCmap& map, 
    SO3partArrayA&, const SO3partArrayA&, const SO3partArrayA&, const hipStream_t&, const int offs, 
    const int mode);




}

#endif 





  /*
  void SO3partA_CGproduct_cu(SO3partArrayA& r, const SO3partArrayA& x, const SO3partArrayA& y, 
    const int mode, const hipStream_t& stream, const int offs){

    const int xl=x.getl();
    const int yl=y.getl();
    const int l=r.getl();

    int Cptr=SO3_cgbank.getfC(xl,yl,l)/4;
    int nlines=x.cellstride/16+y.cellstride/16+r.cellstride/16;
    assert(x.nbu==r.nbu);
    assert(y.nbu==r.nbu);

    const int _nch=1;
    int _nbu=1; if(_nbu<0) _nbu=1;
    nlines/=_nbu;

    if(mode==0){
      dim3 blocks(r.aasize,1,1);
      cnine::CellwiseImap imap;
      SO3partA_CGproduct_kernel<<<blocks,cnine::roundup(x.getn()*y.getn(),32),nlines*128,stream>>>
	(r.arrg,r.arrgc,x.arrg,x.arrgc,y.arrg,y.arrgc,
	  r.cellstride,x.cellstride,y.cellstride,imap,
	  x.getn(),y.getn(),r.getn(),xl,yl,l,offs,_nch,Cptr); // 
    }

    if(mode==1){
      dim3 blocks(x.aasize,y.aasize,1);
      cnine::OuterImap imap(r.adims[1]);
      SO3partA_CGproduct_kernel<<<blocks,cnine::roundup(x.getn()*y.getn(),32),nlines*128,stream>>>
	(r.arrg,r.arrgc,x.arrg,x.arrgc,y.arrg,y.arrgc,
	  r.cellstride,x.cellstride,y.cellstride,imap,
	  x.getn(),y.getn(),r.getn(),xl,yl,l,offs,_nch,Cptr);
    }
    

  }
    */


  /*
  void SO3partA_CGproduct_cu(SO3partArrayA& r, const SO3partArrayA& x, const SO3partArrayA& y, 
    const int rN, const int xN, const int yN, 
    const int ris, const int rjs, const int rks, 
    const int xis, const int xjs, const int xks, 
    const int yis, const int yjs, const int yks, 
    const hipStream_t& stream, const int offs){

    const int xl=x.getl();
    const int yl=y.getl();
    const int l=r.getl();

    int Cptr=SO3_cgbank.getfC(xl,yl,l)/4;
    int nlines=x.cellstride/16+y.cellstride/16+r.cellstride/16;
    assert(x.nbu==r.nbu);
    assert(y.nbu==r.nbu);

    const int _nch=1;
    int _nbu=1; if(_nbu<0) _nbu=1;
    dim3 blocks(rN,xN,yN);
    nlines/=_nbu;


    SO3partA_CGproduct_kernel<<<blocks,cnine::roundup(x.getn()*y.getn(),32),nlines*128,stream>>>
      (r.arrg,r.arrgc,x.arrg,x.arrgc,y.arrg,y.arrgc,
	ris*r.cellstride,rjs*r.cellstride, rks*r.cellstride,
	xis*x.cellstride,xjs*x.cellstride, xks*x.cellstride,
	yis*y.cellstride,yjs*y.cellstride, yks*y.cellstride,
	x.getn(),y.getn(),r.getn(),xl,yl,l,offs,_nch,Cptr); 
  }
  */

/*
__global__ void SO3partA_CGproduct_kernel(float* rarr, float* rarrc, float* xarr, float* xarrc, 
  float* yarr, float* yarrc, 
  const int ristride, const int xistride, const int yistride,   
  const int rjstride, const int xjstride, const int yjstride,   
  const int rkstride, const int xkstride, const int ykstride,   
  const int xfrags, const int yfrags, const int rfrags,  
  const int l1, const int l2, const int l, const int _offs, const int nch, const int Cptr){

  extern __shared__ unsigned char _shared[]; 
  float* shared=reinterpret_cast<float*>(_shared);

  const float* C_ptr=reinterpret_cast<float*>(cg_cmem)+Cptr;

  const int iix=blockIdx.x;
  const int jix=blockIdx.y;
  const int kix=blockIdx.z;

  const int t=threadIdx.x;

  const int r1=2*l1+1;
  const int r2=2*l2+1;
  const int r=2*l+1;

  const int xwidth=xfrags*nch; 
  const int ywidth=yfrags*nch; 
  const int rwidth=xfrags*yfrags*nch;
  const int global_rwidth=rfrags*nch;

  int offs=0;

  int xptr=32*offs;
  SO3part_load(offs,shared,xarr,xarrc,l1,xwidth,iix*xistride+jix*xjstride+kix*xkstride,t);

  const int yptr=32*offs;
  SO3part_load(offs,shared,yarr,yarrc,l2,ywidth,iix*yistride+jix*yjstride+kix*ykstride,t);

  const int rpr=32*offs+t;
  const int rpi=rpr+((r*rwidth-1)/32+1)*32;
  float* _rptr=rarr+iix*ristride+jix*rjstride+kix*rkstride+_offs;
  float* _rptri=rarrc+iix*ristride+jix*rjstride+kix*rkstride+_offs;

  if(t<rwidth){
    for(int i=0; i<r; i++)
      shared[rpr+i*rwidth]=_rptr[i*global_rwidth+t];
    for(int i=0; i<r; i++)
      shared[rpi+i*rwidth]=_rptri[i*global_rwidth+t];
  }

  __syncthreads();
  
  const int xcol=t/yfrags;
  const int xpr=xptr+xcol;
  const int xlines=((r1*xwidth-1)/32+1); 
  const int xpi=xpr+xlines*32;

  const int ycol=t%ywidth;
  const int ypr=yptr+ycol;
  const int ylines=((r2*ywidth-1)/32+1); 
  const int ypi=ypr+ylines*32;

  if(t<rwidth){
    for(int m1=-l1; m1<=l1; m1++){
      const float x_r=shared[xpr+xwidth*(m1+l1)];
      const float x_i=shared[xpi+xwidth*(m1+l1)];
      int lower=-l-m1; if(lower<-l2) lower=-l2;
      int upper=l-m1; if(upper>l2) upper=l2;
      for(int m2=lower; m2<=upper; m2++){
	float c=C_ptr[(m1+l1)*r2+m2+l2];
	const float y_r=shared[ypr+ywidth*(m2+l2)];
	const float y_i=shared[ypi+ywidth*(m2+l2)];
	shared[rpr+rwidth*(m1+m2+l)]+=c*(x_r*y_r-x_i*y_i); 
	shared[rpi+rwidth*(m1+m2+l)]+=c*(x_r*y_i+x_i*y_r);
      }
    }
  }

  __syncthreads();

  if(t<rwidth){
    for(int i=0; i<r; i++)
      _rptr[i*global_rwidth+t]=shared[rpr+i*rwidth];
    for(int i=0; i<r; i++)
      _rptri[i*global_rwidth+t]=shared[rpi+i*rwidth];
  }

}
*/

/*
__device__ int SO3part_load(int& offs, float* shared, float* arr, float* arrc, const int l, const int w, const int skip, const int t){
  const int _offs=offs;
  int ptr=32*offs;
  const int r=2*l+1;
  const int lines=((r*w-1)/32+1); 
  float* xcell=arr+skip; 
  if(t<32)
    for(int i=0; i<lines; i++)
      shared[ptr+i*32+t]=xcell[i*32+t];
  ptr+=32*lines;
  xcell=arrc+skip;
  if(t<32)
    for(int i=0; i<lines; i++)
      shared[ptr+i*32+t]=xcell[i*32+t];
  offs+=2*lines;
  return 32*_offs;
}


__device__ int SO3part_load(int& offs, float* shared, float* arr, float* arrc, const int l, const int w, const int skip, 
  const int frag_offs, const int nfrags, const int t){
  const int _offs=offs;
  int ptr=32*offs+t;
  const int r=2*l+1;
  const int lines=((r*w-1)/32+1); 
  float* xcell=arr+skip+frag_offs+t;
  if(t<nfrags){
    for(int i=0; i<r; i++)
      shared[ptr+i*nfrags]=xcell[i*w];
    ptr+=32*lines;
    xcell=arrc+skip+frag_offs+t;
    for(int i=0; i<r; i++)
      shared[ptr+i*nfrags]=xcell[i*w];
  }
  offs+=2*lines;
  return 32*_offs;
}
*/

