#include "hip/hip_runtime.h"
/*
 * This file is part of GElib, a C++/CUDA library for group equivariant 
 * tensor operations. 
 *  
 * Copyright (c) 2024, Imre Risi Kondor
 *
 * This source code file is subject to the terms of the noncommercial 
 * license distributed with GElib in the file NONCOMMERICAL.TXT. Commercial 
 * use is prohibited. All redistributed versions of this file (in orginal
 * or modified form) must retain this copyright notice and must be 
 * accompanied by a verbatim copy of the license. 
 *
 */

#ifndef _SO3part_addCGproduct_back1_cu
#define _SO3part_addCGproduct_back1_cu

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "SO3_CGbank.hpp"
#include "Ctensor5_view.hpp"
#include "utils.hpp"
#include "utils.cu"


extern GElib::SO3_CGbank SO3_cgbank;


__global__ void SO3part_addCGproduct_back1_tiled_kernel(const cnine::Ctensor5_view x, 
  const cnine::Ctensor4_view r, const cnine::Ctensor5_view y, int xremainder, int yremainder, 
  const int Cptr, float* cptr_global, const bool preloadCG){

  extern __shared__ unsigned char _shared[]; 
  const int b0=blockIdx.x;
  const int b1=blockIdx.y;
  const int t=threadIdx.x;

  int l1=(x.n2-1)/2;
  int l2=(y.n2-1)/2;
  int l=(r.n2-1)/2;
  int L2=y.n2;

  float* cptr;
  float* xpr;
  if(preloadCG){
    cptr=reinterpret_cast<float*>(_shared);
    xpr=cptr+((x.n2*y.n2-1)/32+1)*32;
    if(Cptr>=0) loadf(cptr,reinterpret_cast<float*>(cg_cmem)+Cptr,x.n2*y.n2);
    else loadf(cptr,cptr_global,x.n2*y.n2);
  }else{
    if(Cptr>=0) cptr=reinterpret_cast<float*>(cg_cmem)+Cptr;
    else cptr=cptr_global;
    xpr=reinterpret_cast<float*>(_shared);
  }

  float* xpi=xpr+x.n2*x.n4;
  float* ypr=xpr+((2*x.n2*x.n4-1)/32+1)*32;
  float* ypi=ypr+y.n2*y.n4;

  int xs=x.s2;
  int ys=y.s2;
  int rs=r.s2;
  int ytot=y.n3*y.n4+yremainder;

  for(int j=0; j<=y.n3; j++){
    int yn=y.n4; 
    if(j==y.n3) yn=yremainder;
    if(yn==0) break;
    loadg_tile(ypr,y,j,yn);

    for(int i=0; i<=x.n3; i++){
      int xn=x.n4; 
      if(i==x.n3) xn=xremainder;
      if(xn==0) break;
      loadg_tile(xpr,x,i,xn);

      __syncthreads();

     if(t<yn){
	float* _ypr=ypr+t;
	float* _ypi=ypi+t;
    
	for(int m1=-l1; m1<=l1; m1++){
	  int lower=-l-m1; if(lower<-l2) lower=-l2;
	  int upper=l-m1; if(upper>l2) upper=l2;
	  float y_r=0;
	  float y_i=0;

	  for(int xcol=0; xcol<yn; xcol++){

	    float* _xpr=xpr+xcol;
	    float* _xpi=xpi+xcol;
	    float* _rpr=r.arr+r.s0*b0+r.s1*b1+r.s3*((i*x.n4+t)*ytot+(j*y.n4+ycol));
	    float* _rpi=r.arrc+r.s0*b0+r.s1*b1+r.s3*((i*x.n4+t)*ytot+(j*y.n4+ycol));

	    for(int m2=lower; m2<=upper; m2++){
	      float c=cptr[(m1+l1)*L2+m2+l2];
	      const float x_r=_xpr[xs*(m1+l1)];
	      const float x_i=_xpi[xs*(m1+l1)];
	      const float g_r=_rpr[rs*(m1+m2+l)];
	      const float g_i=_rpi[rs*(m1+m2+l)];
	      y_r+=c*(g_r*x_r+g_i*x_i);
	      y_i+=c*(-g_r*x_i+g_i*x_r);
	    }
	  }

	  _ypr[ys1*(m2+l2)]+=y_r; 
	  _ypi[ys1*(m2+l2)]+=y_i;
	}
     }

    }// for i

    saveg_tile(ypr,y,j,yn);

  }// for j

}


// --------------------------------------------------------------------------------------------------------------------


namespace GElib{


  void SO3part_addCGproduct_back1_cu(SO3part y, SO3part r, SO3part x, const int offs, const hipStream_t& stream){

    GELIB_ASSRT(r.get_dev()==1);
    GELIB_ASSRT(x.get_dev()==1);
    GELIB_ASSRT(y.get_dev()==1);

    const int l1=x.getl();
    const int l2=y.getl();
    const int l=r.getl();
    const int L1=2*l1+1;
    const int L2=2*l2+1;
    GELIB_ASSRT(l>=std::abs(l1-l2) && l<=l1+l2);
    GELIB_ASSRT(r.getn()>=x.getn()*y.getn()+offs);

    r.canonicalize_to_4d();
    x.canonicalize_to_4d();
    y.canonicalize_to_4d();

    const int b=y.getb();
    r.promote_batch_to(b);
    x.promote_batch_to(b);

    const int g=y.getg();
    r.promote_grid_to(g);
    x.promote_grid_to(g);

    int xn=x.getn();
    int yn=cnine::roundup(y.getn(),32)*32;
    int xremainder=tile_channels(x,xn);
    int yremainder=tile_channels(y,yn);

    auto rv=view4_of(r);
    auto xv=view5_of(x);
    auto yv=view5_of(y);

    rv.arr+=rv.s3*offs;
    rv.arrc+=rv.s3*offs;
    //r.n2=x.n2*y.n2;

    float* cptr=nullptr;
    int Cptr=-1; //SO3_cgbank.getfC(xl,yl,l)/4; // const memory switched off for now
    if(Cptr<0) cptr=SO3_CGbank.get<float>(l1,l2,l,r.dev).get_arr();
    int clines=cnine::roundup(L1*L2,32)/32;

    int nlines=cnine::roundup(L1*xn*2,32)/32+
      cnine::roundup(L2*yn*2,32)/32;

    if(nlines<=384){
      bool preloadCG=(nlines+clines<=384);
      dim3 blocks(b,g);
      SO3part_addCGproduct_back1_tiled_kernel<<<blocks,cnine::roundup(yn,32),(nlines+preloadCG*clines)*128,stream>>>
	(yv,rv,xv,xremainder,yremainder,Cptr,cptr,preloadCG);
      return;
    }

    GELIB_ERROR("A single tile of the input and output tensors does not fit in shared memory.")

  }    


}


#endif 
