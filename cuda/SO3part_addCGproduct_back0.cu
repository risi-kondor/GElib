#include "hip/hip_runtime.h"
/*
 * This file is part of GElib, a C++/CUDA library for group equivariant 
 * tensor operations. 
 *  
 * Copyright (c) 2023, Imre Risi Kondor
 *
 * This source code file is subject to the terms of the noncommercial 
 * license distributed with GElib in the file NONCOMMERICAL.TXT. Commercial 
 * use is prohibited. All redistributed versions of this file (in orginal
 * or modified form) must retain this copyright notice and must be 
 * accompanied by a verbatim copy of the license. 
 *
 */

#ifndef _SO3part_addCGproduct_back0_cu
#define _SO3part_addCGproduct_back0_cu

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "SO3CGbank.hpp"
#include "GPUtensor.hpp"
#include "SO3part.hpp"
#include "utils.hpp"
#include "utils.cu"


extern GElib::SO3CGbank SO3_CGbank;
//extern __device__ __constant__ unsigned char cg_cmem[]; 


__global__ void SO3part_addCGproduct_back0_kernel(const cnine::GPUtensor<float,5> x, 
  const cnine::GPUtensor<float,5> r, const cnine::GPUtensor<float,5> y,  
  const int Cptr, float* cptr_global, const bool preloadCG){

  extern __shared__ unsigned char _shared[]; 
  const int b0=blockIdx.x;
  const int b1=blockIdx.y;
  const int b2=blockIdx.z;
  const int t=threadIdx.x;

  int l1=(x.dims[3]-1)/2;
  int l2=(y.dims[3]-1)/2;
  int l=(r.dims[3]-1)/2;
  int L1=x.dims[3];
  int L2=y.dims[3];
  int xn=x.dims[4];
  int yn=y.dims[4];

  float* cptr;
  float* xpr;
  if(preloadCG){
    cptr=reinterpret_cast<float*>(_shared);
    xpr=cptr+((L1*L2-1)/32+1)*32;
    loadf(cptr,cptr_global,L1*L2);
  }else{
    cptr=cptr_global;
    xpr=reinterpret_cast<float*>(_shared);
  }

  float* xpi=xpr+L1*xn;
  float* ypr=xpr+((2*L1*xn-1)/32+1)*32;
  float* ypi=ypr+L2*yn;
  float* rpr=r.arr+r.strides[0]*b0+r.strides[1]*b1+r.strides[2]*b2+t*yn*r.strides[4];

  loadf(xpr,x.arr+x.strides[0]*b0+x.strides[1]*b1+x.strides[2]*b2,L1*xn,x.strides[4]);
  loadf(xpi,x.arr+x.strides[0]*b0+x.strides[1]*b1+x.strides[2]*b2+1,L1*xn,x.strides[4]);

  loadf(ypr,y.arr+y.strides[0]*b0+y.strides[1]*b1+y.strides[2]*b2,L2*yn,y.strides[4]);
  loadf(ypi,y.arr+y.strides[0]*b0+y.strides[1]*b1+y.strides[2]*b2+1,L2*yn,y.strides[4]);

  __syncthreads();

  int xs=xn;
  int ys=yn;
  int rs=r.strides[3];

  if(t<xn){
    float* _xpr=xpr+t;
    float* _xpi=xpi+t;
    
    for(int m1=-l1; m1<=l1; m1++){
      int lower=-l-m1; if(lower<-l2) lower=-l2;
      int upper=l-m1; if(upper>l2) upper=l2;
      float x_r=0;
      float x_i=0;

      for(int ycol=0; ycol<yn; ycol++){

	float* _ypr=ypr+ycol;
	float* _ypi=ypi+ycol;
	float* _rpr=rpr+ycol*r.strides[4];
	float* _rpi=_rpr+1;

	for(int m2=lower; m2<=upper; m2++){
	  float c=cptr[(m1+l1)*L2+m2+l2];
	  const float y_r=_ypr[ys*(m2+l2)];
	  const float y_i=_ypi[ys*(m2+l2)];
	  const float g_r=_rpr[rs*(m1+m2+l)];
	  const float g_i=_rpi[rs*(m1+m2+l)];
	  x_r+=c*(g_r*y_r+g_i*y_i);
	  x_i+=c*(-g_r*y_i+g_i*y_r);
	}
      }
      _xpr[xs*(m1+l1)]+=x_r; 
      _xpi[xs*(m1+l1)]+=x_i;
    }
  }

  savef(x.arr+x.strides[0]*b0+x.strides[1]*b1+x.strides[2]*b2,xpr,L1*xn,x.strides[4]);
  savef(x.arr+x.strides[0]*b0+x.strides[1]*b1+x.strides[2]*b2+1,xpi,L1*xn,x.strides[4]);

}


__global__ void SO3part_addCGproduct_back0_tiled_kernel(const cnine::GPUtensor<float,5> x, 
  const cnine::GPUtensor<float,5> r, const cnine::GPUtensor<float,5> y,  
  int xn, int yn, const int Cptr, float* cptr_global, const bool preloadCG){

  extern __shared__ unsigned char _shared[]; 
  const int b0=blockIdx.x;
  const int b1=blockIdx.y;
  const int b2=blockIdx.z;
  const int t=threadIdx.x;

  int l1=(x.dims[3]-1)/2;
  int l2=(y.dims[3]-1)/2;
  int l=(r.dims[3]-1)/2;
  int L1=x.dims[3];
  int L2=y.dims[3];
  int xN=x.dims[4]/xn;
  int yN=y.dims[4]/yn;

  float* cptr;
  float* xpr;
  if(preloadCG){
    cptr=reinterpret_cast<float*>(_shared);
    xpr=cptr+((L1*L2-1)/32+1)*32;
    loadf(cptr,cptr_global,L1*L2);
  }else{
    cptr=cptr_global;
    xpr=reinterpret_cast<float*>(_shared);
  }

  float* xpi=xpr+L1*xn;
  float* ypr=xpr+((2*L1*xn-1)/32+1)*32;
  float* ypi=ypr+L2*yn;

  float* rarr=r.arr+r.strides[0]*b0+r.strides[1]*b1+r.strides[2]*b2;
  float* xarr=x.arr+x.strides[0]*b0+x.strides[1]*b1+x.strides[2]*b2;
  float* yarr=y.arr+y.strides[0]*b0+y.strides[1]*b1+y.strides[2]*b2;

  int xs=xn;
  int ys=yn;
  int rs=r.strides[3];

  for(int i=0; i<=xN; i++){
    int _xn=xn;
    if(i==xN) _xn=x.dims[4]%xn;
    if(_xn==0) break;

    load_tile(xpr,xarr+i*xn*x.strides[4],L1,_xn,x.strides[3],x.strides[4]);
    load_tile(xpi,xarr+i*xn*x.strides[4]+1,L1,_xn,x.strides[3],x.strides[4]);
    __syncthreads();

    for(int j=0; j<=yN; j++){
      int _yn=yn;
      if(j==yN) _yn=y.dims[4]%yn;
      if(_yn==0) break;

      load_tile(ypr,yarr+j*yn*y.strides[4],L2,_yn,y.strides[3],y.strides[4]);
      load_tile(ypi,yarr+j*yn*y.strides[4]+1,L2,_yn,y.strides[3],y.strides[4]);
      __syncthreads();

      if(t<xn){
	float* _xpr=xpr+t;
	float* _xpi=xpi+t;
    
	for(int m1=-l1; m1<=l1; m1++){
	  int lower=-l-m1; if(lower<-l2) lower=-l2;
	  int upper=l-m1; if(upper>l2) upper=l2;
	  float x_r=0;
	  float x_i=0;

	  for(int ycol=0; ycol<yn; ycol++){

	    float* _ypr=ypr+ycol;
	    float* _ypi=ypi+ycol;
	    float* _rpr=rarr+((i*xn+t)*y.dims[4]+j*yn+ycol)*r.strides[4];
	    float* _rpi=_rpr+1;

	    for(int m2=lower; m2<=upper; m2++){
	      float c=cptr[(m1+l1)*L2+m2+l2];
	      const float y_r=_ypr[ys*(m2+l2)];
	      const float y_i=_ypi[ys*(m2+l2)];
	      const float g_r=_rpr[rs*(m1+m2+l)];
	      const float g_i=_rpi[rs*(m1+m2+l)];
	      x_r+=c*(g_r*y_r+g_i*y_i);
	      x_i+=c*(-g_r*y_i+g_i*y_r);
	    }
	  }
	  _xpr[xs*(m1+l1)]+=x_r; 
	  _xpi[xs*(m1+l1)]+=x_i;
	}
      }

    }// for j

    save_tile(xarr+i*xn*x.strides[4],xpr,L1,_xn,x.strides[3],x.strides[4]);
    save_tile(xarr+i*xn*x.strides[4]+1,xpi,L1,_xn,x.strides[3],x.strides[4]);
    __syncthreads();

  }// for i

}


// --------------------------------------------------------------------------------------------------------------------


namespace GElib{


  void SO3part_addCGproduct_back0_cu(const SO3part<float>& x, const SO3part<float>& r, const SO3part<float>& y, const int offs, const hipStream_t& stream){

    if(r.get_dev()!=1) GELIB_SKIP("SO3part r must be on GPU");
    if(x.get_dev()!=1) GELIB_SKIP("SO3part x must be on GPU");
    if(y.get_dev()!=1) GELIB_SKIP("SO3part y must be on GPU");

    if(r.ndims()!=5) GELIB_SKIP("SO3part r must be 5D");
    if(x.ndims()!=5) GELIB_SKIP("SO3part x must be 5D");
    if(y.ndims()!=5) GELIB_SKIP("SO3part y must be 5D");

    auto rsubdims=r.dims.chunk(0,3);
    if(x.dims.chunk(0,3)!=rsubdims) GELIB_SKIP("leading dimensions of x and r must be same");
    if(y.dims.chunk(0,3)!=rsubdims) GELIB_SKIP("leading dimensions of y and r must be same");
    if(r.dims[0]>65535) GELIB_SKIP("dims[0] exceeds 65535");
    if(r.dims[1]>65535) GELIB_SKIP("dims[1] exceeds 65535");
    if(r.dims[2]>65535) GELIB_SKIP("dims[2] exceeds 65535");
    if((size_t)(r.dims[0])*r.dims[1]*r.dims[2]>INT_MAX) GELIB_SKIP("product of block dimensions exceeds 2^31-1");

    const int l1=x.getl();
    const int l2=y.getl();
    const int l=r.getl();
    const int L1=2*l1+1;
    const int L2=2*l2+1;
    if(l<std::abs(l1-l2) || l>l1+l2) GELIB_SKIP("|l1-l_2| <= l <= l1+l2 not satisfied");

    int xn=x.dims[4];
    int yn=y.dims[4];
    if(xn*yn+offs>r.dims[4]) GELIB_SKIP("fragment dimension of r not large enough");

    cnine::GPUtensor<float,5> rv(r);
    cnine::GPUtensor<float,5> xv(x);
    cnine::GPUtensor<float,5> yv(y);
    rv.arr+=r.strides[4]*offs;

    float* cptr=SO3_CGbank.get<float>(l1,l2,l,r.dev).get_arr(); 
    int clines=cnine::roundup(L1*L2,32)/32;
    int nlines=cnine::roundup(L1*xn*2,32)/32+cnine::roundup(L2*yn*2,32)/32;

    bool tiled=xn>1024;
    if(x.strides[3]!=x.strides[4]*x.dims[4]) tiled=true;
    if(y.strides[3]!=y.strides[4]*y.dims[4]) tiled=true;
    if(nlines+clines>384) tiled=true;

    if(!tiled){
      bool preloadCG=(nlines+clines<=384);
      dim3 blocks(r.dims[0],r.dims[1],r.dims[1]);
      SO3part_addCGproduct_back0_kernel<<<blocks,cnine::roundup(xn,32),(nlines+preloadCG*clines)*128,stream>>>
	(xv,rv,yv,-1,cptr,true);
      return;
    }

    if(tiled){
      xn=std::min(xn,1024);
      int nlines=cnine::roundup(L1*xn*2,32)/32+cnine::roundup(L2*yn*2,32)/32;
      int conservative_clines=clines;
      if(conservative_clines>200) conservative_clines=0; // try preloading CG if it uses at most 200 lines

      if(nlines<=384){ // should always be true
	bool preloadCG=(nlines+clines<=384);
	dim3 blocks(r.dims[0],r.dims[1],r.dims[1]);
	SO3part_addCGproduct_back0_tiled_kernel<<<blocks,cnine::roundup(xn,32),(nlines+preloadCG*clines)*128,stream>>>
	  (xv,rv,yv,xn,yn,-1,cptr,true);
	return;
      }

    }


    GELIB_SKIP("tiled kernel not supported.");

  }    


}


#endif 
