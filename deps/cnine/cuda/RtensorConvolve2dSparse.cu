#include "hip/hip_runtime.h"
/*
 * This file is part of cnine, a lightweight C++ tensor library. 
 *  
 * Copyright (c) 2021, Imre Risi Kondor
 *
 * This source code file is subject to the terms of the noncommercial 
 * license distributed with cnine in the file LICENSE.TXT. Commercial 
 * use is prohibited. All redistributed versions of this file (in 
 * original or modified form) must retain this copyright notice and 
 * must be accompanied by a verbatim copy of the license. 
 *
 */

#ifndef _RtensorConvolveSparse_cu
#define _RtensorConvolveSparse_cu

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/complex.h>
#include <thrust/tuple.h>

#include "Cnine_base.hpp"
#include "Rtensor2_view.hpp"
#include "Rtensor3_view.hpp"
#include "Rtensor5_view.hpp"
#include "Itensor1_view.hpp"
#include "Itensor2_view.hpp"
#include "CSRmatrix.hpp"
#include "CUDAhelpers.hpp"


__global__ void RtensorConvolve2d_sparse_kernel
(float* rarr, const int rs0, const int rs1, const int rs2, const int rs3, const int rs4, 
  float* xarr, const int xs0, const int xs1, const int xs2, const int xs3, const int xs4,  
  float* warr, int* wdir, const int rn1, const int nj1, const int na){

  int i0=blockIdx.y/rn1;
  int i1=blockIdx.y%rn1;

  int row=blockIdx.y*blockDim.z+blockIdx.z;
  int offs=wdir[2*row];
  int n=wdir[2*row+1];
  
  float t=0;
  for(int i=0; i<n; i++){
    int s=*reinterpret_cast<int*>(warr+offs+2*i);
    int j0=s/(nj1*na);
    int j1=(s/na)%nj1;
    int a=s%na;
    t+=xarr[blockIdx.x*xs0+(i0+j0)*xs1+(i1+j1)*xs2+a*xs3+threadIdx.x*xs4]*warr[offs+2*i+1];
  }
  rarr[blockIdx.x*rs0+i0*rs1+i1*rs2+blockIdx.z*rs3+threadIdx.x*rs4]+=t;
  
}


__global__ void RtensorConvolve2d_sparse_kernel
(float* rarr, const int rs0, const int rs1, const int rs2, const int rs3, const int rs4, 
  float* xarr, const int xs0, const int xs1, const int xs2, const int xs3, const int xs4,  
  float* warr, int* wdir, const int rn1, const int nj1, const int na,
  const int xn1, const int xn2, const int padding0, const int padding1){

  int i0=blockIdx.y/rn1;
  int i1=blockIdx.y%rn1;

  int row=blockIdx.y*blockDim.z+blockIdx.z;
  int offs=wdir[2*row];
  int n=wdir[2*row+1];
  
  float t=0;
  for(int i=0; i<n; i++){
    int s=*reinterpret_cast<int*>(warr+offs+2*i);
    int j0=s/(nj1*na);
    int j1=(s/na)%nj1;
    if(i0+j0-padding0<0 || i0+j0-padding0>=xn1) continue;
    if(i1+j1-padding1<0 || i1+j1-padding1>=xn2) continue;
    int a=s%na;
    t+=xarr[blockIdx.x*xs0+(i0+j0-padding0)*xs1+(i1+j1-padding1)*xs2+a*xs3+threadIdx.x*xs4]*warr[offs+2*i+1];
  }
  rarr[blockIdx.x*rs0+i0*rs1+i1*rs2+blockIdx.z*rs3+threadIdx.x*rs4]+=t;
  
}


// ----------------------------------------------------------------------------------------------------------
// ----------------------------------------------------------------------------------------------------------
// ----------------------------------------------------------------------------------------------------------


namespace cnine{


  void RtensorConvolve2d_cu(const Rtensor5_view& r, const Rtensor5_view& x, const CSRmatrix<float>& w, 
    const int J0, const int J1, const int padding0, const int padding1, const hipStream_t& stream){
    CNINE_ASSRT(r.dev==1);
    CNINE_ASSRT(x.dev==1);
    CNINE_ASSRT(w.dev==1);

    dim3 blocks(r.n0,r.n1*r.n2,r.n3);

    if(padding0==0&&padding1==0){
      RtensorConvolve2d_sparse_kernel<<<blocks,r.n4,0,stream>>>
	(r.arr,r.s0,r.s1,r.s2,r.s3,r.s4,
	  x.arr,x.s0,x.s1,x.s2,x.s3,x.s4,
	  w.arrg,w.get_dirg(1),
	  J0,J1,w.m); // check this
    }else{
      //RtensorConvolve2d_sparse_kernel<<<blocks,r.n4,0,stream>>>
      //(r.arr,r.s0,r.s1,r.s2,r.s3,r.s4,
      //  x.arr,x.s0,x.s1,x.s2,x.s3,x.s4,
      //  w.arrg,w.get_dirg(1),
      //  r.n1,w.n2,w.m,
      //  x.n1,x.n2,padding0,padding1); 
    }

  }

}


#endif 
