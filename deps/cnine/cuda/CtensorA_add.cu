#include "hip/hip_runtime.h"
/*
 * This file is part of cnine, a lightweight C++ tensor library. 
 *  
 * Copyright (c) 2021, Imre Risi Kondor
 *
 * This source code file is subject to the terms of the noncommercial 
 * license distributed with cnine in the file LICENSE.TXT. Commercial 
 * use is prohibited. All redistributed versions of this file (in 
 * original or modified form) must retain this copyright notice and 
 * must be accompanied by a verbatim copy of the license. 
 *
 */

#ifndef _CtensorA_add_cu
#define _CtensorA_add_cu

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/complex.h>
#include <thrust/tuple.h>

#include "Cmaps.hpp"
#include "CtensorArrayA.hpp"


template<typename IMAP>
__global__ void CtensorA_add_kernel(float* rarr, float* rarrc, float* xarr, float* xarrc, 
  const int rstride, const int xstride, const IMAP& map){
  
  auto T=map(blockIdx.x,blockIdx.y,blockIdx.z);
  const int rix=thrust::get<0>(T);
  const int xix=thrust::get<1>(T);
  const int t=threadIdx.x;

  rarr[rix*rstride+t]+=xarr[xix*xstride+t];
  rarrc[rix*rstride+t]+=xarrc[xix*xstride+t];
}


namespace cnine{

  template<typename CMAP>
  void CtensorA_add_cu(const CMAP& map, CtensorArrayA& r, const CtensorArrayA& x, const hipStream_t& stream){

    
    CtensorA_add_kernel<<<map.blockdims(),r.asize,0,stream>>>
      (r.arrg,r.arrgc,x.arrg,x.arrgc,r.cellstride,x.cellstride,map);
    
  }

  template void CtensorA_add_cu<cnine::CellwiseUCmap>(const cnine::CellwiseUCmap& map, 
    CtensorArrayA&, const CtensorArrayA&, const hipStream_t&);

  template void CtensorA_add_cu<cnine::BroadcastUCmap>(const cnine::BroadcastUCmap& map, 
    CtensorArrayA&, const CtensorArrayA&, const hipStream_t&);

}

#endif 
